#include "hip/hip_runtime.h"
#include "solver_cuda/CudaSolver.hpp"

__global__ void makeStepKernel(
        real_type *dst, const real_type *fprev, const real_type *fcur,
        unsigned int width, unsigned int height,
        ModelParameters modelParameters,
        SolverParameters solverParameters)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < width && j < height) {
        auto i_plus = i==width-1? 0: i+1;
        auto i_minus = i==0? width-1: i-1;
        auto j_plus = j==height-1? 0: j+1;
        auto j_minus = j==0? height-1: j-1;

        real_type gamma = modelParameters.getWaveSpeed()*
                solverParameters.getTimeStepLength()/solverParameters.getSpatialStepLength();
        real_type lapl =
                fcur[i_plus+j*width] +
                fcur[i+j_plus*width] +
                fcur[i_minus+j*width]+
                fcur[i+j_minus*width]-
                4*fcur[i+j*width];
        real_type u=2*fcur[i+j*width]-fprev[i+j*width]+gamma*gamma*lapl;
        dst[i+j*width] = u;
    }
}

class CudaSolver::Impl
{
public:
    void makeStep(
        const ModelParameters& modelParameters,
        const SolverParameters& solverParameters,
        const CudaSolverDataFrame& fprev,
        const CudaSolverDataFrame& fcur,
        CudaSolverDataFrame& fnext)
    {
        const unsigned int BLOCK_SIZE = 16;
        auto dst = fnext.data();
        dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
        dim3 dimGrid((fprev.width()+dimBlock.x-1) / dimBlock.x, (fprev.height()+dimBlock.y-1) / dimBlock.y, 1);

        makeStepKernel<<<dimGrid, dimBlock>>> (
            thrust::raw_pointer_cast<real_type*>(fnext.deviceData().data()),
            thrust::raw_pointer_cast<const real_type*>(fprev.deviceData().data()),
            thrust::raw_pointer_cast<const real_type*>(fcur.deviceData().data()),
            fnext.width(), fnext.height(),
            modelParameters, solverParameters);
    }
};

CudaSolver::CudaSolver() :
    m_impl(std::make_shared<Impl>())
{
}

void CudaSolver::makeStep(
        const ModelParameters& modelParameters,
        const SolverParameters& solverParameters,
        const CudaSolverDataFrame& fprev,
        const CudaSolverDataFrame& fcur,
        CudaSolverDataFrame& fnext)
{
    m_impl->makeStep(
        modelParameters,
        solverParameters,
        fprev,
        fcur,
        fnext);
}
