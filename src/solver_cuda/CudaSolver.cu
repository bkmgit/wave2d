#include "hip/hip_runtime.h"
#include "solver_cuda/CudaSolver.hpp"

__global__ void makeStepKernel(
        real_type *dst, const real_type *fprev, const real_type *fcur,
        unsigned int width, unsigned int height,
        ModelParameters modelParameters,
        SolverParameters solverParameters)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < width && j < height) {
        auto i_plus = i==width-1? 0: i+1;
        auto i_minus = i==0? width-1: i-1;
        auto j_plus = j==height-1? 0: j+1;
        auto j_minus = j==0? height-1: j-1;

        real_type gamma = modelParameters.getWaveSpeed()*
                solverParameters.getTimeStepLength()/solverParameters.getSpatialStepLength();
        real_type lapl =
                fcur[i_plus+j*width] +
                fcur[i+j_plus*width] +
                fcur[i_minus+j*width]+
                fcur[i+j_minus*width]-
                4*fcur[i+j*width];
        real_type u=2*fcur[i+j*width]-fprev[i+j*width]+gamma*gamma*lapl;
        dst[i+j*width] = u;
    }
}

class CudaSolver::Impl
{
public:
    ~Impl() {
         maybeFree();
    }
    void makeStep(
        const ModelParameters& modelParameters,
        const SolverParameters& solverParameters,
        const DataFrame& fprev,
        const DataFrame& fcur,
        DataFrame& fnext)
{
    const unsigned int BLOCK_SIZE = 16;
    auto dst = fnext.data();
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
    dim3 dimGrid((fprev.width()+dimBlock.x-1) / dimBlock.x, (fprev.height()+dimBlock.y-1) / dimBlock.y, 1);

    auto size = fnext.width()*fnext.height()*sizeof(real_type);
    maybeAlloc(size);

    hipMemcpy(m_fprev, fprev.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(m_fcur, fcur.data(), size, hipMemcpyHostToDevice);

    makeStepKernel<<<dimGrid, dimBlock>>> (
        m_fnext, m_fprev, m_fcur,
        fnext.width(), fnext.height(),
        modelParameters, solverParameters);

    hipMemcpy(fnext.data(), m_fnext, size, hipMemcpyDeviceToHost);
}

private:
    real_type *m_fprev = nullptr;
    real_type *m_fcur = nullptr;
    real_type *m_fnext = nullptr;

    void maybeAlloc(size_t size) {
        if (!m_fprev) {
            hipMalloc(&m_fprev, size);
            hipMalloc(&m_fcur, size);
            hipMalloc(&m_fnext, size);
        }
    }

    void maybeFree()
    {
        if (m_fprev) {
            hipFree(m_fprev);
            hipFree(m_fcur);
            hipFree(m_fnext);
            m_fprev = nullptr;
        }
    }

};

CudaSolver::CudaSolver() :
    m_impl(std::make_shared<Impl>())
{
}

void CudaSolver::makeStep(
        const ModelParameters& modelParameters,
        const SolverParameters& solverParameters,
        const DataFrame& fprev,
        const DataFrame& fcur,
        DataFrame& fnext)
{
    m_impl->makeStep(
        modelParameters,
        solverParameters,
        fprev,
        fcur,
        fnext);
}
