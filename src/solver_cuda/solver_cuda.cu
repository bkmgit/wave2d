#include "hip/hip_runtime.h"
#include "real_type.hpp"
#include "DataFrame.hpp"
#include "ModelParameters.hpp"
#include "SolverParameters.hpp"
#include "SolverInterface.hpp"

__global__ void makeStepKernel()
{
}

class CudaSolver : public SolverInterface
{
public:
    void makeStep(
            const ModelParameters& modelParameters,
            const SolverParameters& solverParameters,
            const DataFrame& fprev,
            const DataFrame& fcur,
            DataFrame& fnext)
    {
	// TODO
	const unsigned int BLOCK_SIZE = 16;
        auto dst = fnext.data();
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
	dim3 dimGrid((fprev.width()+dimBlock.x-1) / dimBlock.x, (fprev.height()+dimBlock.y-1) / dimBlock.y, 1);
    
	makeStepKernel<<<dimGrid, dimBlock>>> ();
    }
};
